
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <omp.h>

char *cp_to_device(char *from, size_t size);
void cp_to_host(char *to, char*from, size_t size);
__global__ void ccc_loop1(const int * __restrict imaterial, const int * __restrict nextfrac, const double * __restrict rho_compact, const double * __restrict rho_compact_list, 
						  const double * __restrict  Vf_compact_list, const double * __restrict  V, double * __restrict rho_ave_compact, int sizex, int sizey, int * __restrict mmc_index) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if (i >= sizex || j >= sizey) return;
#ifdef FUSED
	double ave = 0.0;
	int ix = imaterial[i+sizex*j];

	if (ix <= 0) {
		// condition is 'ix >= 0', this is the equivalent of
		// 'until ix < 0' from the paper
#ifdef LINKED
		for (ix = -ix; ix >= 0; ix = nextfrac[ix]) {
			ave += rho_compact_list[ix] * Vf_compact_list[ix];
		}
#else
		for (int idx = mmc_index[-ix]; idx < mmc_index[-ix+1]; idx++) {
			ave += rho_compact_list[idx] * Vf_compact_list[idx];	
		}
#endif
		rho_ave_compact[i+sizex*j] = ave/V[i+sizex*j];
	}
	else {
#endif
		// We use a distinct output array for averages.
		// In case of a pure cell, the average density equals to the total.
		rho_ave_compact[i+sizex*j] = rho_compact[i+sizex*j] / V[i+sizex*j];
#ifdef FUSED
	}
#endif
}


__global__ void ccc_loop1_2(const double * __restrict rho_compact_list, const double * __restrict  Vf_compact_list, const double * __restrict  V, double * __restrict rho_ave_compact, const int * __restrict mmc_index,  const int  mmc_cells,  const int * __restrict mmc_i,  const int * __restrict mmc_j, int sizex, int sizey) {
	int c = threadIdx.x + blockIdx.x * blockDim.x;
	if (c >= mmc_cells) return;
   double ave = 0.0;
   for (int m = mmc_index[c]; m < mmc_index[c+1]; m++) {
     ave +=  rho_compact_list[m] * Vf_compact_list[m];
   }
   rho_ave_compact[mmc_i[c]+sizex*mmc_j[c]] = ave/V[mmc_i[c]+sizex*mmc_j[c]];
}

__global__ void ccc_loop2(const int * __restrict imaterial, const int * __restrict matids, const int * __restrict nextfrac, const double * __restrict rho_compact, const double * __restrict rho_compact_list, 
						const double * __restrict t_compact, const double * __restrict t_compact_list, 
						  const double * __restrict  Vf_compact_list, const double * __restrict n, double * __restrict  p_compact, double * __restrict p_compact_list, int sizex, int sizey, int * __restrict mmc_index) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if (i >= sizex || j >= sizey) return;

	int ix = imaterial[i+sizex*j];
	if (ix <= 0) {
#ifdef FUSED
		// NOTE: I think the paper describes this algorithm (Alg. 9) wrong.
		// The solution below is what I believe to good.

		// condition is 'ix >= 0', this is the equivalent of
		// 'until ix < 0' from the paper
#ifdef LINKED
		for (ix = -ix; ix >= 0; ix = nextfrac[ix]) {
			double nm = n[matids[ix]];
			p_compact_list[ix] = (nm * rho_compact_list[ix] * t_compact_list[ix]) / Vf_compact_list[ix];
		}
#else
		for (int idx = mmc_index[-ix]; idx < mmc_index[-ix+1]; idx++) {
			double nm = n[matids[idx]];
			p_compact_list[idx] = (nm * rho_compact_list[idx] * t_compact_list[idx]) / Vf_compact_list[idx];
		}
#endif
#endif
	}
	else {
		// NOTE: HACK: we index materials from zero, but zero can be a list index
		int mat = ix - 1;
		// NOTE: There is no division by Vf here, because the fractional volume is 1.0 in the pure cell case.
		p_compact[i+sizex*j] = n[mat] * rho_compact[i+sizex*j] * t_compact[i+sizex*j];;
	}
}
__global__ void ccc_loop2_2(const int * __restrict matids, const double * __restrict rho_compact_list, 
						const double * __restrict t_compact_list, const double * __restrict  Vf_compact_list, const double * __restrict n, double * __restrict p_compact_list, int * __restrict mmc_index, int mmc_cells) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= mmc_cells) return;
  double nm = n[matids[idx]];
  p_compact_list[idx] = (nm * rho_compact_list[idx] * t_compact_list[idx]) / Vf_compact_list[idx];
}

__global__ void ccc_loop3(const int * __restrict imaterial, const int * __restrict nextfrac, const int * __restrict matids,
							 double * __restrict rho_compact, 
							 double * __restrict rho_compact_list, const double * __restrict x, const double * __restrict y,
						  int sizex, int sizey, int * __restrict mmc_index) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if (i >= sizex-1 || j >= sizey-1 || i < 1 || j < 1) return;


	// o: outer
	double xo = x[i+sizex*j];
	double yo = y[i+sizex*j];

	// There are at most 9 neighbours in 2D case.
	double dsqr[9];

	// for all neighbours
	for (int nj = -1; nj <= 1; nj++) {

		for (int ni = -1; ni <= 1; ni++) {

			dsqr[(nj+1)*3 + (ni+1)] = 0.0;

			// i: inner
			double xi = x[(i+ni)+sizex*(j+nj)];
			double yi = y[(i+ni)+sizex*(j+nj)];

			dsqr[(nj+1)*3 + (ni+1)] += (xo - xi) * (xo - xi);
			dsqr[(nj+1)*3 + (ni+1)] += (yo - yi) * (yo - yi);
		}
	}

	int ix = imaterial[i+sizex*j];

	if (ix <= 0) {
		// condition is 'ix >= 0', this is the equivalent of
		// 'until ix < 0' from the paper
		#ifdef LINKED
		for (ix = -ix; ix >= 0; ix = nextfrac[ix]) {
		#else
		for (int ix = mmc_index[-imaterial[i+sizex*j]]; ix < mmc_index[-imaterial[i+sizex*j]+1]; ix++) {
		#endif
			int mat = matids[ix];
			double rho_sum = 0.0;
			int Nn = 0;

			// for all neighbours
			for (int nj = -1; nj <= 1; nj++) {

				for (int ni = -1; ni <= 1; ni++) {

					int ci = i+ni, cj = j+nj;
					int jx = imaterial[ci+sizex*cj];

					if (jx <= 0) {
						// condition is 'jx >= 0', this is the equivalent of
						// 'until jx < 0' from the paper
						#ifdef LINKED
						for (jx = -jx; jx >= 0; jx = nextfrac[jx]) {
						#else
						for (int jx = mmc_index[-imaterial[ci+sizex*cj]]; jx < mmc_index[-imaterial[ci+sizex*cj]+1]; jx++) {
						#endif
							if (matids[jx] == mat) {
								rho_sum += rho_compact_list[jx] / dsqr[(nj+1)*3 + (ni+1)];
								Nn += 1;

								// The loop has an extra condition: "and not found".
								// This makes sense, if the material is found, there won't be any more of the same.
								break;
							}
						}
					}
					else {
						// NOTE: In this case, the neighbour is a pure cell, its material index is in jx.
						// In contrast, Algorithm 10 loads matids[jx] which I think is wrong.

						// NOTE: HACK: we index materials from zero, but zero can be a list index
						int mat_neighbour = jx - 1;
						if (mat == mat_neighbour) {
							rho_sum += rho_compact[ci+sizex*cj] / dsqr[(nj+1)*3 + (ni+1)];
							Nn += 1;
						}
					} // end if (jx <= 0)
				} // end for (int ni)
			} // end for (int nj)

			rho_compact_list[ix] = rho_sum / Nn;
		} // end for (ix = -ix)
	} // end if (ix <= 0)
	else {
		// NOTE: In this case, the cell is a pure cell, its material index is in ix.
		// In contrast, Algorithm 10 loads matids[ix] which I think is wrong.

		// NOTE: HACK: we index materials from zero, but zero can be a list index
		int mat = ix - 1;

		double rho_sum = 0.0;
		int Nn = 0;

		// for all neighbours
		for (int nj = -1; nj <= 1; nj++) {
			if ((j + nj < 0) || (j + nj >= sizey)) // TODO: better way?
				continue;

			for (int ni = -1; ni <= 1; ni++) {
				if ((i + ni < 0) || (i + ni >= sizex)) // TODO: better way?
					continue;

				int ci = i+ni, cj = j+nj;
				int jx = imaterial[ci+sizex*cj];

				if (jx <= 0) {
					// condition is 'jx >= 0', this is the equivalent of
					// 'until jx < 0' from the paper
					#ifdef LINKED
					for (jx = -jx; jx >= 0; jx = nextfrac[jx]) {
					#else
					for (int jx = mmc_index[-imaterial[ci+sizex*cj]]; jx < mmc_index[-imaterial[ci+sizex*cj]+1]; jx++) {
					#endif
						if (matids[jx] == mat) {
							rho_sum += rho_compact_list[jx] / dsqr[(nj+1)*3 + (ni+1)];
							Nn += 1;

							// The loop has an extra condition: "and not found".
							// This makes sense, if the material is found, there won't be any more of the same.
							break;
						}
					}
				}
				else {
					// NOTE: In this case, the neighbour is a pure cell, its material index is in jx.
					// In contrast, Algorithm 10 loads matids[jx] which I think is wrong.

					// NOTE: HACK: we index materials from zero, but zero can be a list index
					int mat_neighbour = jx - 1;
					if (mat == mat_neighbour) {
						rho_sum += rho_compact[ci+sizex*cj] / dsqr[(nj+1)*3 + (ni+1)];
						Nn += 1;
					}
				} // end if (jx <= 0)
			} // end for (int ni)
		} // end for (int nj)

		rho_compact[i+sizex*j] = rho_sum / Nn;
	} // end else

}

void compact_cell_centric(int sizex, int sizey, int Nmats,
	int *imaterial, int *matids, int *nextfrac,
	double *x, double *y, double *n,
	double *rho_compact, double *rho_compact_list, double *rho_ave_compact,
	double *p_compact, double *p_compact_list,
	double *t_compact, double *t_compact_list,
	double *V, double *Vf_compact_list, int mm_len, int mmc_cells, int *mmc_index, int *mmc_i, int *mmc_j)
{

	int    *d_imaterial = (int *)cp_to_device((char*)imaterial, sizex*sizey*Nmats*sizeof(int));
	int    *d_matids = (int *)cp_to_device((char*)matids, mm_len*sizeof(int));
	int    *d_nextfrac = (int *)cp_to_device((char*)nextfrac, mm_len*sizeof(int));
	int    *d_mmc_index = (int *)cp_to_device((char*)mmc_index, (mmc_cells+1)*sizeof(int));
	int    *d_mmc_i = (int *)cp_to_device((char*)mmc_i, (mmc_cells)*sizeof(int));
	int    *d_mmc_j = (int *)cp_to_device((char*)mmc_j, (mmc_cells)*sizeof(int));
	double *d_x = (double *)cp_to_device((char*)x, sizex*sizey*sizeof(double));
	double *d_y = (double *)cp_to_device((char*)y, sizex*sizey*sizeof(double));
	double *d_rho_compact = (double *)cp_to_device((char*)rho_compact, sizex*sizey*sizeof(double));
	double *d_rho_compact_list = (double *)cp_to_device((char*)rho_compact_list,mm_len*sizeof(double));
	double *d_p_compact = (double *)cp_to_device((char*)p_compact, sizex*sizey*sizeof(double));
	double *d_p_compact_list = (double *)cp_to_device((char*)p_compact_list,mm_len*sizeof(double));
	double *d_t_compact = (double *)cp_to_device((char*)t_compact, sizex*sizey*sizeof(double));
	double *d_t_compact_list = (double *)cp_to_device((char*)t_compact_list,mm_len*sizeof(double));
	double *d_Vf_compact_list = (double *)cp_to_device((char*)Vf_compact_list, mm_len*sizeof(double));
	double *d_V = (double *)cp_to_device((char*)V, sizex*sizey*sizeof(double));
	double *d_n = (double *)cp_to_device((char*)n, Nmats*sizeof(double));
	double *d_rho_ave_compact = (double *)cp_to_device((char*)rho_ave_compact, sizex*sizey*sizeof(double));

	int thx = 32;
	int thy = 4;
	dim3 threads(thx,thy,1);
	dim3 blocks((sizex-1)/thx+1, (sizey-1)/thy+1, 1);


	// Cell-centric algorithms
	// Computational loop 1 - average density in cell
  double t1 = omp_get_wtime();
  ccc_loop1<<<blocks, threads>>>(d_imaterial, d_nextfrac, d_rho_compact, d_rho_compact_list, d_Vf_compact_list, d_V, d_rho_ave_compact, sizex, sizey, d_mmc_index);
#ifndef FUSED
  ccc_loop1_2<<<(mmc_cells-1)/(thx*thy)+1, (thx*thy)>>>(d_rho_compact_list, d_Vf_compact_list, d_V, d_rho_ave_compact, d_mmc_index, mmc_cells, d_mmc_i, d_mmc_j, sizex, sizey);
#endif
  hipDeviceSynchronize();
  printf("Compact matrix, cell centric, alg 1: %g sec\n", omp_get_wtime()-t1);
	// Computational loop 2 - Pressure for each cell and each material
  t1 = omp_get_wtime();
  ccc_loop2<<<blocks, threads>>>(d_imaterial, d_matids,d_nextfrac, d_rho_compact, d_rho_compact_list, d_t_compact, d_t_compact_list, d_Vf_compact_list, d_n, d_p_compact, d_p_compact_list, sizex, sizey, d_mmc_index);
#ifndef FUSED
  ccc_loop2_2<<<(mm_len-1)/(thx*thy)+1, (thx*thy)>>>(d_matids, d_rho_compact_list, d_t_compact_list, d_Vf_compact_list, d_n, d_p_compact_list, d_mmc_index, mm_len);
#endif
  hipDeviceSynchronize();
  printf("Compact matrix, cell centric, alg 2: %g sec\n", omp_get_wtime()-t1);

	// Computational loop 3 - Average density of each material over neighborhood of each cell
  t1 = omp_get_wtime();
	ccc_loop3<<<blocks, threads>>>(d_imaterial,d_nextfrac, d_matids, d_rho_compact, d_rho_compact_list, d_x, d_y, sizex, sizey, d_mmc_index);  
  hipDeviceSynchronize();
 	printf("Compact matrix, cell centric, alg 3: %g sec\n", omp_get_wtime()-t1);
  
	cp_to_host((char*)x, (char*)d_x, sizex*sizey*sizeof(double));
	cp_to_host((char*)y, (char*)d_y, sizex*sizey*sizeof(double));
	cp_to_host((char*)rho_compact, (char*)d_rho_compact, sizex*sizey*sizeof(double));
	cp_to_host((char*)rho_compact_list, (char*)d_rho_compact_list, mm_len*sizeof(double));
	cp_to_host((char*)p_compact, (char*)d_p_compact, sizex*sizey*sizeof(double));
	cp_to_host((char*)p_compact_list, (char*)d_p_compact_list, mm_len*sizeof(double));
	cp_to_host((char*)t_compact, (char*)d_t_compact, sizex*sizey*sizeof(double));
	cp_to_host((char*)t_compact_list, (char*)d_t_compact_list, mm_len*sizeof(double));
	cp_to_host((char*)Vf_compact_list, (char*)d_Vf_compact_list, mm_len*sizeof(double));
	cp_to_host((char*)V, (char*)d_V, sizex*sizey*sizeof(double));
	cp_to_host((char*)n, (char*)d_n, Nmats*sizeof(double));
	cp_to_host((char*)rho_ave_compact, (char*)d_rho_ave_compact, sizex*sizey*sizeof(double));
}

bool compact_check_results(int sizex, int sizey, int Nmats,
	int *imaterial, int *matids, int *nextfrac,
	double *rho_ave, double *rho_ave_compact,
	double *p, double *p_compact, double *p_compact_list,
	double *rho, double *rho_compact, double *rho_compact_list, int *mmc_index)
{
	printf("Checking results of compact representation... ");

	for (int j = 0; j < sizey; j++) {
		for (int i = 0; i < sizex; i++) {
			if (abs(rho_ave[i+sizex*j] - rho_ave_compact[i+sizex*j]) > 0.0001) {
				printf("1. full matrix and compact cell-centric values are not equal! (%f, %f, %d, %d)\n",
					rho_ave[i+sizex*j], rho_ave_compact[i+sizex*j], i, j);
				return false;
			}
			int ix = imaterial[i+sizex*j];
			if (ix <= 0) {
#ifdef LINKED
				for (ix = -ix; ix >= 0; ix = nextfrac[ix]) {
#else
        for (int ix = mmc_index[-imaterial[i+sizex*j]]; ix < mmc_index[-imaterial[i+sizex*j]+1]; ix++) {
#endif
					int mat = matids[ix];
					if (abs(p[(i+sizex*j)*Nmats+mat] - p_compact_list[ix]) > 0.0001) {
						printf("2. full matrix and compact cell-centric values are not equal! (%f, %f, %d, %d, %d)\n",
							p[(i+sizex*j)*Nmats+mat], p_compact_list[ix], i, j, mat);
						return false;
					}

					if (abs(rho[(i+sizex*j)*Nmats+mat] - rho_compact_list[ix]) > 0.0001) {
						printf("3. full matrix and compact cell-centric values are not equal! (%f, %f, %d, %d, %d)\n",
							rho[(i+sizex*j)*Nmats+mat], rho_compact_list[ix], i, j, mat);
						return false;
					}
				}
			}
			else {
				// NOTE: HACK: we index materials from zero, but zero can be a list index
				int mat = ix - 1;
				if (abs(p[(i+sizex*j)*Nmats+mat] - p_compact[i+sizex*j]) > 0.0001) {
					printf("2. full matrix and compact cell-centric values are not equal! (%f, %f, %d, %d, %d)\n",
						p[(i+sizex*j)*Nmats+mat], p_compact[i+sizex*j], i, j, mat);
					return false;
				}

				if (abs(rho[(i+sizex*j)*Nmats+mat] - rho_compact[i+sizex*j]) > 0.0001) {
					printf("3. full matrix and compact cell-centric values are not equal! (%f, %f, %d, %d, %d)\n",
						rho[(i+sizex*j)*Nmats+mat], rho_compact[i+sizex*j], i, j, mat);
					return false;
				}
      }
    }
  }
	printf("All tests passed!\n");
	return true;
}
